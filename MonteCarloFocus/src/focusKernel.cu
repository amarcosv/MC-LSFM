#include "hip/hip_runtime.h"
#include "focusKernel.h"
#include "focus_utils.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_occupancy.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#define CUDA_ERROR_CHECK
#ifdef CUDA_DEBUG
#define PHOTON_DEBUG(x) print_photon(x);
#define DEBUG(x) printf x;
#else
#define PHOTON_DEBUG(x) 
#define DEBUG(x) 
#endif


#define PRINT(fp,...) fprintf(fp,__VA_ARGS__)

__constant__ focusConfig fcfg[1];
const int LOAD_BUFFER_SIZE = 2097152;
//const int LOAD_BUFFER_SIZE = 3;
/**
*@brief Error Handler for Cuda Api calls
*/
#define CUDA_ASSERT( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}
#endif

	return;
}

/**
*@brief Error Handler for Kernel calls
*/
#define CUDA_ERROR_CHECK()    __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaCheckError(const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}

	// More careful checking. However, this will affect performance.
	// Comment away if needed.
	err = hipDeviceSynchronize();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}
#endif

	return;
}

/**
*@brief Copy Array to GPU for Kernel calls
*arguments: (**device,*host,number of elements, data type)
*/
#define CUDA_LOAD(device,host,elements,type) CopyArrayToGPU((void**)device,(void*)host,elements, sizeof(type))
int CopyArrayToGPU(void** DeviceArray, void* HostArray, unsigned long int NumElements, size_t typelength)
{
    unsigned long int bytes = typelength * NumElements;

    // Allocate memory on the GPU for array
    if (hipMalloc(DeviceArray, bytes) != hipSuccess)
    {
	printf("CopyArrayToGPU(): Couldn't allocate mem for array on GPU.");
	return 1;
    }

    // Copy the contents of the host array to the GPU
    if (hipMemcpy(*DeviceArray, HostArray, bytes, hipMemcpyHostToDevice) != hipSuccess)
    {
	printf("CopyArrayToGPU(): Couldn't copy host array to GPU.");
	hipFree(*DeviceArray);
	return 1;
    }

    return 0;
}


/**
*@brief Copy GPU Array to Host for Kernel calls
*arguments: (**host,*device,elements, type)
*/
#define CUDA_UNLOAD(host,device,elements,type) CopyArrayToHost((void**)host,(void*)device,elements, sizeof(type))
int CopyArrayToHost(void **HostArray, void *DeviceArray, unsigned long int NumElements, size_t typelength)
{
	unsigned long int bytes = typelength * NumElements;

	// Copy the contents of the host array to the GPU
	if (hipMemcpy(*HostArray, DeviceArray, bytes, hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("CopyArrayToGPU(): Couldn't copy device array to host.");
		hipFree(*HostArray);
		return 1;
	}

	return 0;
}


/**
*@brief Build focus settings structure to be copied in constant memory
*/
void createfocusConfig(SPIMConfig* cfg, SPIMGPUInfo* gpu, focusConfig* fcfg) {
	fcfg->f1 = cfg->f1/cfg->mcx_pxSize; 
	fcfg->f2 = cfg->f2 /cfg->mcx_pxSize;
	fcfg->NA1 = cfg->NA1;
	fcfg->NA2 = cfg->NA2;
		//fcfg->lens1R = cfg->mcx_simVolume.x / 2.f ;
	//fcfg->lens2R = fcfg->lens1R;
	fcfg->lens1R = fcfg->f1 * tanf(asinf(0.99f * fcfg->NA1));
	fcfg->lens2R = fcfg->f2 * tanf(asinf(0.99f * fcfg->NA2));
	fcfg->sensorFactor = cfg->sensorpxSize/cfg->mcx_pxSize;
	fcfg->sensorSize.x = cfg->sensorSize.x;
	fcfg->sensorSize.y = cfg->sensorSize.y;
	fcfg->focusZplane=cfg->zScan.x / cfg->mcx_pxSize; /** Convert to Grid units*/
	fcfg->mcx_simVolume.x = cfg->mcx_simVolume.x;
	fcfg->mcx_simVolume.y = cfg->mcx_simVolume.y;
	fcfg->mcx_simVolume.z = cfg->mcx_simVolume.z;
	fcfg->mcx_pxSize = cfg->mcx_pxSize;
	fcfg->ill_simVolume.x = cfg->ill_simVolume.x;
	fcfg->ill_simVolume.y = cfg->ill_simVolume.y;
	fcfg->ill_simVolume.z = cfg->ill_simVolume.z;
	fcfg->ill_pxSize = cfg->ill_pxSize;
	fcfg->Nphotons = cfg->Nphotons;
	fcfg->blockSize = gpu->autoblock;   /**< The launch configurator returned block size*/
	fcfg->gridSize = gpu->autothread / fcfg->blockSize;    /**< The actual grid size needed, based on input size*/
	fcfg->photonsPerThread = (unsigned long int)ceil((float)cfg->Nphotons / (float)gpu->autothread);	
	fcfg->photonSize = cfg->photonSize;
	fcfg->mua = cfg->mua;

	if (cfg->illVolume) {
		float *gillVolume;
		//if (cfg->ill_simVolume.z!=1)
		//CUDA_LOAD(&gillVolume, cfg->illVolume, cfg->ill_simVolume.x * cfg->ill_simVolume.y * (2 * cfg->ill_simVolume.z + 1));
		//else
		    CUDA_LOAD(&gillVolume, cfg->illVolume, cfg->ill_simVolume.x * cfg->ill_simVolume.y * cfg->ill_simVolume.z,float);
		fcfg->illVolume = gillVolume;
		//fcfg->illVolume = NULL;
	}


}
#define FOCUS_AT(x,y) set_focusDistance<<<1,1>>>(x, y); hipDeviceSynchronize();
/**
*@brief Sets the focal plane z distance for the focusing kernel
*
*@param[in] focusZplane:  focal plane in grid units
*@param[in] focusZinfex:  index at output volume to store image
*/
__global__ void set_focusDistance(float focusZplane, int focusZindex) {
	fcfg->focusZplane = focusZplane;
	fcfg->focusZindex = focusZindex;
}

#define SET_PHOTONS_DATA(x,y,z) set_photons<<<1,1>>>(x, y, z); hipDeviceSynchronize();
/**
*@brief Sets the focal plane z distance for the focusing kernel [d in grid units]
*/
__global__ void set_photons(unsigned int photons, float mcx_pxSize, int photonSize) {
	fcfg->Nphotons = photons;
	fcfg->mcx_pxSize = mcx_pxSize;
	fcfg->photonSize = photonSize;
}

/**
*@brief prints photon position information
*/
__global__ void print_photonPosition(photon *photons, unsigned long index) {
	printf("Photon %d: pos[%f, %f, %f]\n", index, photons[index].pos.x, photons[index].pos.y, photons[index].pos.z);
}

__global__ void print_focusConfigGPU() {
	printf("Focus Configuration:\n\tFocus1=%f\n\tFocus2=%f\n\tNphotons=%d\n", fcfg->f1, fcfg->f2, fcfg->Nphotons);
	printf("\tmua=%f\n\tFocus2=%d\n\tPhperthread=%d\n", fcfg->mua, fcfg->photonSize, fcfg->photonsPerThread);
	return;
}



#define PHOTON_PRINT(ph,id) print_photonData<<<1,1>>>(ph,id); hipDeviceSynchronize();
/**
*@brief prints photon position information
*/
__global__ void print_photonData(photon *photons,  unsigned long index) {
	printf("Photon %d:\n", index);
	printf("\t pos: [%f, %f, %f]\n", photons[index].pos.x, photons[index].pos.y, photons[index].pos.z);
	printf("\t dir: [%f, %f, %f]\n", photons[index].dir.x, photons[index].dir.y, photons[index].dir.z);
	printf("\t pos0: [%f, %f, %f]\n", photons[index].Opos.x, photons[index].Opos.y, photons[index].Opos.z);
	printf("\t w:	 %e\n",  photons[index].w);
	
}

/**
*@brief prints photon position information
*/
__device__ void print_photon(photon *photons) {
	printf("\t pos: [%f, %f, %f]\n", photons->pos.x, photons->pos.y, photons->pos.z);
	printf("\t dir: [%f, %f, %f]\n", photons->dir.x, photons->dir.y, photons->dir.z);
	printf("\t w:	 %e\n", photons->w);

}

/**
*@brief Calculates photon weight and shifts photon coordinates to have the (0,0) at the center of the volume 
*
* @param[in, out] photons:  photon data structure
*
*/
__global__ void prepare_MCXphotonData(photon* photons) {
    size_t thread = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread < fcfg->Nphotons) {
	photons[thread].pos.x = photons[thread].pos.x - fcfg->mcx_simVolume.x / 2.0f;
	photons[thread].pos.y = photons[thread].pos.y - fcfg->mcx_simVolume.y / 2.0f;
	photons[thread].Opos.x = photons[thread].Opos.x - fcfg->mcx_simVolume.x / 2.0f;
	photons[thread].Opos.y = photons[thread].Opos.y - fcfg->mcx_simVolume.y / 2.0f;
	photons[thread].w = expf(-photons[thread].w  * fcfg->mua); /** Both are in grid units!*/
    }

}


/**
*@brief Sets photon data in photons array from the raw photonsArray
*
*@param[in,out] photons: structure of photon data
*@param[in]	photonsData: gpu buffer with raw photons info [id ppath(M) p(3) v(3) p0(3) path]
*@param[in]	Nphotons:	 number of photons to be processed
*@param[in]	loadOffset:	 offset
*
*/
__global__ void set_photonData(photon* photons, float* photonsData, unsigned int Nphotons, int offset) {
	size_t thread = blockIdx.x * blockDim.x + threadIdx.x;
	int limit = thread * fcfg->photonsPerThread + fcfg->photonsPerThread;
	if (limit > Nphotons) limit = Nphotons ;
	photons = photons + offset;

	for (int i = thread * fcfg->photonsPerThread; i < limit; i++) {
		//printf("Thread# %d: processing photonData %d into photon %d\n", (int)thread, i, i * fcfg->photonSize);
		photons[i].pos.x = photonsData[i * fcfg->photonSize + 3] -fcfg->mcx_simVolume.x / 2.0f;
		photons[i].pos.y = photonsData[i * fcfg->photonSize + 4] -fcfg->mcx_simVolume.y / 2.0f;
		photons[i].pos.z = photonsData[i * fcfg->photonSize + 5];
		photons[i].dir.x = photonsData[i * fcfg->photonSize + 6];
		photons[i].dir.y = photonsData[i * fcfg->photonSize + 7];
		photons[i].dir.z = photonsData[i * fcfg->photonSize + 8];
		photons[i].Opos.x = photonsData[i * fcfg->photonSize + 9];
		photons[i].Opos.y = photonsData[i * fcfg->photonSize + 10];
		photons[i].Opos.z = photonsData[i * fcfg->photonSize + 11];
		photons[i].w = expf(-photonsData[i * fcfg->photonSize + 2]*fcfg->mcx_pxSize * fcfg->mua);
		//photons[i].w = photonsData[i * fcfg->photonSize + 2] * 0.025f;
	}
	return;
}

/**
*@brief Sets position element in photons array from the raw photonsArray
*/
__global__ void set_photonPosition(photon *photons, float *photonsData, unsigned long int Nphotons) {
	size_t thread = blockIdx.x * blockDim.x + threadIdx.x;	
	size_t limit = thread * fcfg->photonsPerThread + fcfg->photonsPerThread;
	if (limit > Nphotons) limit = Nphotons; /* Prevent from accesing array out of bounds photons*/
	for (int i = thread * fcfg->photonsPerThread; i < limit; i++) {
		photons[i].pos.x = photonsData[i] - fcfg->mcx_simVolume.x / 2.0f;
		photons[i].pos.y = photonsData[i + Nphotons] - fcfg->mcx_simVolume.y / 2.0f;
		photons[i].pos.z = photonsData[i + 2 * Nphotons];
	}
	return;
}

/**
*@brief Sets position element in photons array from the raw photonsArray
*/
/**__global__ void set_photonPosition(photon *photons,float *photonsData, unsigned long int Nphotons){
	size_t thread = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread < Nphotons) {
		photons[thread].pos.x = photonsData[thread] - fcfg->mcx_simVolume.x/2.0f;
		photons[thread].pos.y = photonsData[thread + Nphotons] - fcfg->mcx_simVolume.y / 2.0f;
		photons[thread].pos.z = photonsData[thread + 2 * Nphotons] ;
	}
	return;
}*/
/**
*@brief Sets direction element in photons array from the raw photonsArray
*/
__global__ void set_photonDirection(photon *photons, float *photonsData, unsigned long int Nphotons) {
	size_t thread = blockIdx.x * blockDim.x + threadIdx.x;	
	size_t limit = thread * fcfg->photonsPerThread + fcfg->photonsPerThread;
	if (limit > Nphotons) limit = Nphotons;
	for (int i = thread * fcfg->photonsPerThread; i < limit; i++) {
		photons[i].dir.x = photonsData[i];
		photons[i].dir.y = photonsData[i + 1 * Nphotons];
		photons[i].dir.z = photonsData[i + 2 * Nphotons];
	}
	return;
}

/**
*@brief Sets launch position in photons array from the raw photonsArray
*/
__global__ void set_photonOrigin(photon0Pos *photonsOrigin, float *photonsData, unsigned long int Nphotons) {
	size_t thread = blockIdx.x * blockDim.x + threadIdx.x;	
	size_t limit = thread * fcfg->photonsPerThread + fcfg->photonsPerThread;
	if (limit > Nphotons) limit = Nphotons;
	for (int i = thread * fcfg->photonsPerThread; i < limit; i++) {
		photonsOrigin[i].x = __float2int_rn(photonsData[i]);
		photonsOrigin[i].y = __float2int_rn(photonsData[i + 1 * Nphotons]);
		photonsOrigin[i].z = __float2int_rn(photonsData[i + 2 * Nphotons]);
	}
	return;
}

/**
*@brief Sets photonWeight value in photons array from the raw photonsArray
*/
__global__ void set_photonWeight(photon *photons, float *photonsData, unsigned long int Nphotons) {
	size_t thread = blockIdx.x * blockDim.x + threadIdx.x;	
	size_t limit = thread * fcfg->photonsPerThread + fcfg->photonsPerThread;
	if (limit > Nphotons) limit = Nphotons;
	for (int i = thread * fcfg->photonsPerThread; i < limit; i++) {
		photons[i].w = photonsData[i];
	}
	return;
}



/**
*@brief Builds the photonsStruct using the raw data from the .mch binary file.
*/
int load_photonStruct(focusConfig* hfcfg, SPIMGPUInfo* gpu, unsigned long int Nphotons, float* photonsData, photon** photons) {
	float* gphotonsData = NULL;	/**< Device buffer to store temporally raw photon data*/
	/*Prepare GPU settings*/
	int blockSize = gpu->autoblock;   /**< The launch configurator returned block size*/
	int gridSize = gpu->autothread/gpu->autoblock;    /**< The actual grid size needed, based on input size*/
	int nChunks = Nphotons / LOAD_BUFFER_SIZE;
	int remaining= Nphotons % LOAD_BUFFER_SIZE;
	int loadBuffer = LOAD_BUFFER_SIZE;
	if (remaining > 0) ++nChunks;

	printf("Running memory allocation kernel for %d photons.\nAllocation in %d steps of %d photons per chunk\n", Nphotons, nChunks, LOAD_BUFFER_SIZE);	
	/*Allocate photons structures*/
	CUDA_ASSERT(hipMalloc((void**)photons, Nphotons * sizeof(photon)));
	//CUDA_ASSERT(hipMalloc((void**)photonsOrigin, Nphotons * sizeof(photon0Pos)));
	CUDA_ASSERT(hipMalloc(&gphotonsData, LOAD_BUFFER_SIZE * hfcfg->photonSize * sizeof(float)));
	//blockSize = 64;
		//gridSize = 2560;

	int offset = 0;
	int step = 0;
	while (offset < Nphotons) {
		if (offset + LOAD_BUFFER_SIZE > Nphotons) loadBuffer = Nphotons - offset;
		printf("\tStep %d of %d: Loading photons  %d to %d.\n", step+1,nChunks,offset, offset+ loadBuffer);
		CUDA_ASSERT(hipMemcpy(gphotonsData, photonsData + offset * hfcfg->photonSize, loadBuffer * hfcfg->photonSize * sizeof(float), hipMemcpyHostToDevice));
		set_photonData << <gridSize, blockSize >> > (*photons, gphotonsData, loadBuffer, offset);
		hipDeviceSynchronize();
		CUDA_ERROR_CHECK();
		offset += LOAD_BUFFER_SIZE;
		step++;
	}

	CUDA_ASSERT(hipFree(gphotonsData));
	return 0;
}







/**
*@brief Builds the photonsStruct using the raw data from the binary file. 
*/
int build_photonStruct(unsigned long int Nphotons, float *photonsData, photon **photons, photon0Pos **photonsOrigin){
	float* gphotonsData = NULL;	/**< Device buffer to store temporally raw photon data*/ 
	/*Prepare GPU settings*/
	int blockSize=128;   /**< The launch configurator returned block size*/ 
	int gridSize=12288/blockSize;    /**< The actual grid size needed, based on input size*/ 
	
	//gridSize = (Nphotons + blockSize - 1) / blockSize;
	printf("Running memory allocation kernel for %d photons. %d blocks of %d threads per block\n", Nphotons, gridSize, blockSize);

	/*Allocate photons structure*/
	CUDA_ASSERT(hipMalloc((void **)photons, Nphotons*sizeof(photon)));
	CUDA_ASSERT(hipMalloc((void **)photonsOrigin, Nphotons*sizeof(photon0Pos)));


	/*Assign photons position*/
	CUDA_LOAD(&gphotonsData, photonsData, 3 * Nphotons, float);
	set_photonPosition << <gridSize, blockSize >> > (*photons, gphotonsData,Nphotons);
	hipDeviceSynchronize();
	CUDA_ERROR_CHECK();

	/*Assign photons direction*/
	CUDA_LOAD(&gphotonsData, (photonsData + 3 * Nphotons), 3 * Nphotons, float);
	set_photonDirection << <gridSize, blockSize >> > (*photons, gphotonsData, Nphotons);
	hipDeviceSynchronize();
	CUDA_ERROR_CHECK();

	/*Assign photons launch position*/
	CUDA_LOAD(&gphotonsData, (photonsData + 6 * Nphotons), 3 * Nphotons,float);
	set_photonOrigin << <gridSize, blockSize >> > (*photonsOrigin, gphotonsData, Nphotons);
	hipDeviceSynchronize();
	CUDA_ERROR_CHECK();

	CUDA_ASSERT(hipFree(gphotonsData));
	/*Assign photons launch position*/
	CUDA_LOAD(&gphotonsData, (photonsData + 9 * Nphotons), Nphotons, float);
	set_photonWeight << <gridSize, blockSize >> > (*photons, gphotonsData, Nphotons);
	hipDeviceSynchronize();
	CUDA_ERROR_CHECK();


	CUDA_ASSERT(hipFree(gphotonsData));
	return 0;
}

/**
*@brief Build camera detector buffer.
*/
int build_cameraSensor(SPIMConfig *cfg, float **cameraSensor) {
	CUDA_ASSERT(hipMalloc((void **)cameraSensor, (cfg->sensorSize.x * cfg->sensorSize.y) * cfg->nzPlanes *  sizeof(float)));
	CUDA_ASSERT(hipMemset(*cameraSensor, 0, (cfg->sensorSize.x * cfg->sensorSize.y) * cfg->nzPlanes * sizeof(float)));
	return 0;
}

/**
*@brief Propagates photon a distance z
*/
__device__ void photonProp(photon* photon, float z) {
	float t = fdividef(z , photon->dir.z);
	photon->pos.x += t * photon->dir.x;
	photon->pos.y += t * photon->dir.y;
}

/**
*@brief Refract photon according to the lens focal distance
*/
__device__ void photonRefract(photon* photon, float f) {

	photonDir dirP;		/** photon direction in photon's coordinate system*/
	photonDir dirPr;	/** rotated photon direction in photon's coordinate system*/
	float2 P;
	float h;	
	float sintheta;
	float costheta;
	/**Normalize photon base vector*/	
	h = rsqrtf(photon->pos.x*photon->pos.x + photon->pos.y*photon->pos.y);
	P.x = photon->pos.x*h;
	P.y = photon->pos.y*h;	
	sincosf(1.0f/(f*h),&sintheta,&costheta);	
	/** Change photon coordinate system*/
	dirP.x = photon->dir.x*P.x + photon->dir.y*P.y;
	dirP.y = photon->dir.y*P.x - photon->dir.x*P.y;
	dirP.z = photon->dir.z;	
	/** Rotate photon*/
	dirPr.x = dirP.x*costheta + dirP.z*sintheta;
	dirPr.y = dirP.y;
	dirPr.z = -dirP.x*sintheta + dirP.z*costheta;	
	/**Revert to system, coordinate system*/
	photon->dir.x = P.x*dirPr.x - P.y*dirPr.y;
	photon->dir.y = P.y*dirPr.x + P.x*dirPr.y;
	photon->dir.z = dirPr.z;

}

/**
*@brief Check if a photon passes through a puppil
*True if photon passes through the lens 
*/
__device__ int checkPupil(photon* photon, float r) {
	if ((photon->pos.x*photon->pos.x + photon->pos.y*photon->pos.y) > (r*r))
		return 0;
	return 1;
}

/**
*@brief Check if a photon passes through a puppil
*True if photon passes through the lens
*/
__device__ int checkLensAcceptance(photon* photon, float NA) {
	float mod = sqrtf(photon->dir.x* photon->dir.x+ photon->dir.y* photon->dir.y+ photon->dir.z* photon->dir.z);
	float sintheta = sinf(acosf(photon->dir.z/mod));
	if (sintheta > NA)
		return 0;
	return 1;
}

/**
*@brief Detect a photon hitting the camera sensor
*/
__device__ int detectPhoton(photon* photon, float* cameraSensor) {
	uint2 cameraPixel;
	float2 sensorLandPos;
	int weightIndex;
	int weightOffset;
	/** Convert to detector units*/
	sensorLandPos.x = (photon->pos.x )/fcfg->sensorFactor + fdividef(fcfg->sensorSize.x, 2.0f);
	sensorLandPos.y = (photon->pos.y )/fcfg->sensorFactor + fdividef(fcfg->sensorSize.y, 2.0f);
	DEBUG(("\tLanding at [%f %f]", sensorLandPos.x, sensorLandPos.y));
	/** Check detector out of bounds*/
	if ((sensorLandPos.x < 0.f )|| (sensorLandPos.y < 0.f))
		return 0;
	cameraPixel.x = __float2uint_rn(sensorLandPos.x);
	cameraPixel.y = __float2uint_rn(sensorLandPos.y);
	if ((cameraPixel.x > (fcfg->sensorSize.x-(unsigned int)1)) || (cameraPixel.y > (fcfg->sensorSize.y-(unsigned int)1)))
		return 0;
	
	/** Add weight*/
	DEBUG(("\t\tDetected at [%d %d] IDX: %d\n", cameraPixel.x, cameraPixel.y, cameraPixel.y*fcfg->sensorSize.x + cameraPixel.x));
	//unsigned long volumeIndex = cameraPixel.y * fcfg->sensorSize.x + cameraPixel.x + fcfg->focusZindex * fcfg->sensorSize.x * fcfg->sensorSize.y;


		//***********************
		//weightIndex = photon->Opos.z * fcfg->mcx_simVolume.x * fcfg->mcx_simVolume.y + photon->Opos.y * fcfg->mcx_simVolume.x + photon->Opos.x;
		//weightOffset = ((fcfg->mcx_simVolume.z-1) - __float2uint_rn(fcfg->focusZplane)) * fcfg->mcx_simVolume.x * fcfg->mcx_simVolume.y;
		//atomicAdd(&cameraSensor[cameraPixel.x + cameraPixel.y * fcfg->sensorSize.x + fcfg->focusZindex * fcfg->sensorSize.x * fcfg->sensorSize.y], photon->w * fcfg->illVolume[weightIndex + weightOffset]);

		//***********************

		unsigned int ill_focusZplane = __float2uint_rn(fcfg->focusZplane * fdividef(fcfg->mcx_pxSize,fcfg->ill_pxSize));
		weightOffset = ((fcfg->ill_simVolume.z /2) - ill_focusZplane) * fcfg->ill_simVolume.x * fcfg->ill_simVolume.y;

		uint3 ill_0pos;
		ill_0pos.x = __float2uint_rn(photon->Opos.x * fdividef(fcfg->mcx_pxSize, fcfg->ill_pxSize));		
		ill_0pos.z = __float2uint_rn(photon->Opos.z * fdividef(fcfg->mcx_pxSize, fcfg->ill_pxSize));

		if (fcfg->ill_simVolume.y != 1)
		    ill_0pos.y = __float2uint_rn(photon->Opos.y * fdividef(fcfg->mcx_pxSize, fcfg->ill_pxSize));
		else 
		    ill_0pos.y = 0;

		//weightIndex = photon->Opos.z * fcfg->ill_simVolume.x * fcfg->ill_simVolume.y + photon->Opos.y * fcfg->ill_simVolume.x + photon->Opos.x;
		weightIndex = ill_0pos.z * fcfg->ill_simVolume.x * fcfg->ill_simVolume.y + ill_0pos.y * fcfg->ill_simVolume.x + ill_0pos.x;
		//printf("Photon launched from[%f %f %f] -> [%u %u %u]\n Focus at %f Offset= %d Index= %d ILLIDX= %d\n", photon->Opos.x, photon->Opos.y, photon->Opos.z, ill_0pos.x, ill_0pos.y, ill_0pos.z, fcfg->focusZplane, weightOffset,weightIndex, weightIndex + weightOffset);
		//DEBUG(("\t\tPhoton launched from [%f %f %f] IDX: %d\n", photon->Opos.x, photon->Opos.y, photon->Opos.z,cameraPixel.y * fcfg->sensorSize.x + cameraPixel.x));
		if ((weightIndex + weightOffset) >= 0 && (weightIndex + weightOffset) < (fcfg->ill_simVolume.x * fcfg->ill_simVolume.y * fcfg->ill_simVolume.z)) {
		    atomicAdd(&cameraSensor[cameraPixel.x + cameraPixel.y * fcfg->sensorSize.x + fcfg->focusZindex * fcfg->sensorSize.x * fcfg->sensorSize.y], photon->w * fcfg->illVolume[weightIndex + weightOffset]);
		    return 1;
		}
		//***********************
	return 0;
}
/**
*@brief Focusing Kernel for 4F system
*/
__global__ void focus4F(photon *photons, float* cameraSensor) {
	size_t thread = blockIdx.x * blockDim.x + threadIdx.x;
	
	size_t limit = thread * fcfg->photonsPerThread + fcfg->photonsPerThread;
	
	if (limit > fcfg->Nphotons) limit = fcfg->Nphotons;
	for (int i = thread * fcfg->photonsPerThread; i < limit; i++) {
		//printf("focusing Photon # [%d]", i);
		photon threadphoton = photons[i];
		DEBUG(("focusing at %f  px\n", fcfg->focusZplane));
		DEBUG(("Photon #%d\t", thread));
		PHOTON_DEBUG(&threadphoton);
		/**Propagate to Lens 1 */
		//photonProp(&threadphoton, fcfg->f1 - fcfg->focusZplane);
		photonProp(&threadphoton, fcfg->f1 - (fcfg->mcx_simVolume.z - fcfg->focusZplane));
		DEBUG(("Photon #%d\t", thread));
		PHOTON_DEBUG(&threadphoton);
		/** Lens 1 refraction*/
		if (checkPupil(&threadphoton, fcfg->lens1R) == 0) continue;
		if(checkLensAcceptance(&threadphoton, fcfg->NA1)==0) continue;
		photonRefract(&threadphoton, -fcfg->f1);
		DEBUG(("Photon #%d\t", thread));
		PHOTON_DEBUG(&threadphoton);
		/**Propagate to Lens 2 */
		photonProp(&threadphoton, fcfg->f1 + fcfg->f2);
		DEBUG(("Photon #%d\t", thread));
		PHOTON_DEBUG(&threadphoton);
		/** Lens 2 refraction*/
		if (checkPupil(&threadphoton, fcfg->lens2R) == 0) continue;
		photonRefract(&threadphoton, -fcfg->f2);
		DEBUG(("Photon #%d\t", thread));
		PHOTON_DEBUG(&threadphoton);
		/** Propagate to detector */
		photonProp(&threadphoton, fcfg->f2);
		DEBUG(("Photon #%d\t", thread));
		PHOTON_DEBUG(&threadphoton);
		/** Detect photon*/
		DEBUG(("Photon #%d\t", thread));
		detectPhoton(&threadphoton, cameraSensor);

	}
}

/**
*@brief Focusing Kernel for 4F system
*/
__global__ void focus4F_auto(photon* photons, float* cameraSensor,unsigned int* gfocused) {
    size_t thread = blockIdx.x * blockDim.x + threadIdx.x;
    int res;
   // size_t limit = thread * fcfg->photonsPerThread + fcfg->photonsPerThread;
    if (thread < fcfg->Nphotons) {	
	    //printf("focusing Photon # [%d]", i);
	    photon threadphoton = photons[thread];
	    DEBUG(("focusing at %f  px\n", fcfg->focusZplane));
	    DEBUG(("Photon #%d\t", thread));
	    PHOTON_DEBUG(&threadphoton);
	    /**Propagate to Lens 1 */
	    //photonProp(&threadphoton, fcfg->f1 - fcfg->focusZplane);
	    photonProp(&threadphoton, fcfg->f1 - (fcfg->mcx_simVolume.z - fcfg->focusZplane));
	    DEBUG(("Photon #%d\t", thread));
	    PHOTON_DEBUG(&threadphoton);
	    /** Lens 1 refraction*/
	    if (checkPupil(&threadphoton, fcfg->lens1R) == 0) return;
	    if (checkLensAcceptance(&threadphoton, fcfg->NA1) == 0) return;
	    photonRefract(&threadphoton, -fcfg->f1);
	    DEBUG(("Photon #%d\t", thread));
	    PHOTON_DEBUG(&threadphoton);
	    /**Propagate to Lens 2 */
	    photonProp(&threadphoton, fcfg->f1 + fcfg->f2);
	    DEBUG(("Photon #%d\t", thread));
	    PHOTON_DEBUG(&threadphoton);
	    /** Lens 2 refraction*/
	    if (checkPupil(&threadphoton, fcfg->lens2R) == 0) return;
	    photonRefract(&threadphoton, -fcfg->f2);
	    DEBUG(("Photon #%d\t", thread));
	    PHOTON_DEBUG(&threadphoton);
	    /** Propagate to detector */
	    photonProp(&threadphoton, fcfg->f2);
	    DEBUG(("Photon #%d\t", thread));
	    PHOTON_DEBUG(&threadphoton);
	    /** Detect photon*/
	    DEBUG(("Photon #%d\t", thread));
	    res= detectPhoton(&threadphoton, cameraSensor);
	    atomicAdd(gfocused+fcfg->focusZindex, (unsigned int)res);
    }
}

__global__ void addingKernel(float * imageVolume, unsigned int zPlanes) {
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread < fcfg->sensorSize.x * fcfg->sensorSize.y) {
		for (int i = 1; i < zPlanes; i++) 
			imageVolume[thread] = imageVolume[thread] + imageVolume[thread + fcfg->sensorSize.x * fcfg->sensorSize.y * i];
	}
}

void volumeAdd(SPIMConfig*cfg, SPIMGPUInfo* gpu, float* imageVolume) {
	int blockSize = 1024;   /**< The launch configurator returned block size*/
	int gridSize = (gpu->autothread / gpu->autoblock);    /**< The actual grid size needed, based on input size*/
	gridSize = (cfg->sensorSize.x * cfg->sensorSize.y) / blockSize+1;
	printf("Generating OPT projection...\n");
	addingKernel << <gridSize, blockSize >> > (imageVolume,  cfg->nzPlanes);
	hipDeviceSynchronize();
}


__global__ void output_addingKernel(float* outputVolume, float* imageVolume,int npixels) {
    int thread = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread < npixels) {
	outputVolume[thread] = outputVolume[thread] + imageVolume[thread];
    }
}


/**
*@brief Initializes focus config structure from MCX parameters
*/
void update_outputVolume(SPIMConfig* cfg, SPIMGPUInfo* gpu, float** outputVolume, float* imagevolume) {
    int blockSize = 1024;   /**< The launch configurator returned block size*/
    int gridSize = (cfg->sensorSize.x * cfg->sensorSize.y * cfg->nzPlanes) / blockSize + 1;
    float* goutputVolume;
    float* gimageVolume;
    printf("Updating output volume..\n");
    CUDA_LOAD(&goutputVolume, *outputVolume,cfg->sensorSize.x * cfg->sensorSize.y * cfg->nzPlanes,float);
    CUDA_LOAD(&gimageVolume, imagevolume, cfg->sensorSize.x * cfg->sensorSize.y * cfg->nzPlanes,float);

    output_addingKernel << <gridSize, blockSize >> > (goutputVolume, gimageVolume, cfg->sensorSize.x * cfg->sensorSize.y * cfg->nzPlanes);
    CUDA_ASSERT(hipDeviceSynchronize());
    CUDA_UNLOAD(outputVolume, goutputVolume, cfg->sensorSize.x * cfg->sensorSize.y * cfg->nzPlanes,float);
    CUDA_ASSERT(hipFree(goutputVolume));
    CUDA_ASSERT(hipFree(gimageVolume));

}





__global__ void illTest() {
	printf("This is a test of illumination volume\n");
	for (int i = 0; i < 10; i++) {
		printf("\t Pos %d: %f\n", i, fcfg->illVolume[i]);
	}
}

/**
*@brief check card memory and estimate tunning configuration 
*/
 void memoryCalculator(focusConfig* hfcfg, int *nChunks, unsigned int *photonsPerChunk, unsigned int *remaindingPhotons) {
	size_t cudaMemory;			/**  Cuda total device memory*/
	size_t cudaFreeMemory;		/**  Cuda device free memory*/
	size_t cudaFreememory_ph;	/**  Cuda device free memory in photons*/
	

	/*Check  memory needs*/
	CUDA_ASSERT(hipMemGetInfo(&cudaFreeMemory, &cudaMemory));
	printf("Photons struct size in GPU = %zu KB = %.2f GB\n", hfcfg->Nphotons * sizeof(photon), ((float)hfcfg->Nphotons * sizeof(photon)) / 1073741824.0f);
	printf("Free memory space in GPU = %d KB = %.2f GB of %.2f GB\n", cudaFreeMemory, cudaFreeMemory / 1073741824.0f, cudaMemory / 1073741824.0f);
	cudaFreememory_ph = (cudaFreeMemory / sizeof(photon)) * 0.8;

	/*Estimate memory needs*/
	*nChunks = (int)hfcfg->Nphotons / cudaFreememory_ph;
	*remaindingPhotons = (unsigned int)hfcfg->Nphotons % cudaFreememory_ph;
	if (remaindingPhotons > 0) ++(*nChunks);
	*photonsPerChunk = cudaFreememory_ph;

	printf("Processing %d photons in %u steps of %d photons. Remainding photons = %u\n",hfcfg->Nphotons,*nChunks,*photonsPerChunk, *remaindingPhotons);
	
}
 /**
*@brief Print CPU focusing Kernel
*/
 void print_focusConfig(SPIMConfig* cfg) {
	 printf("Focus Configuration:\n\tf1 = %f [mm]\n\tf2 = %f [mm]\n", cfg->f1, cfg->f2);
	 printf("\tmua = %f [cm^-1]\n\t Number of .mch files = %z\n", cfg->mua, cfg->nFiles);
	 return;
 }


int focus_launcher(SPIMConfig* cfg, SPIMGPUInfo* gpu, float** imageVolume) {
	focusConfig hfcfg;
	photon* photons;
	photon0Pos* photonsOrigin;
	float* photonsData = NULL;
	int nChunks;
	unsigned int photonsPerChunk, remainingPhotons;
	size_t cudaMemory;
	size_t cudaFreeMemory;
	float* gimageVolume;
	float zFocus;
	/* Prepare GPU settings*/
	int blockSize = gpu->autoblock;   /**< The launch configurator returned block size*/
	int gridSize = gpu->autothread / blockSize;    /**< The actual grid size needed, based on input size*/
	/* Prepare variables to import data files*/
	//int fidx = 0;
	char tmp[200];
	const char* dataFile;
	int Nphotons;

	CUDA_ASSERT(hipMemGetInfo(&cudaFreeMemory, &cudaMemory));
	printf("Free memory space in GPU = %d KB = %.2f GB of %.2f GB\n", cudaFreeMemory, cudaFreeMemory / 1073741824.0f, cudaMemory / 1073741824.0f);

	/*Prepare camera buffer*/
	build_cameraSensor(cfg, &gimageVolume);

	/*Generate kernel focusing settings*/
	createfocusConfig(cfg, gpu, &hfcfg);
	hfcfg.photonsPerThread = (unsigned long int)ceil((float)LOAD_BUFFER_SIZE / (float)gpu->autothread);
	CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(fcfg), &hfcfg, sizeof(focusConfig)));

	/*print_focusConfig << <1, 1 >> > ();
	hipDeviceSynchronize();*/
	   
	int photonOffset;

	for (int fidx = 0; fidx < cfg->nFiles; fidx++) {
		printf("Processing file %d of %d \n", fidx + 1, cfg->nFiles);
		/*Load photons data from file*/
		buildMchPath(cfg, tmp, fidx);
		dataFile = &(*tmp);
		Nphotons = readMchFile(dataFile, cfg, &photonsData);	
		//Nphotons = 100;
		if (Nphotons > 0) {
			printf("Data was loaded succesfully!!\n");
			cfg->Nphotons += (unsigned int)Nphotons;

			/*Update focusing kernel structure*/
			hfcfg.Nphotons = (unsigned long int)Nphotons;
			hfcfg.mcx_pxSize = cfg->mcx_pxSize;
			hfcfg.photonSize = cfg->photonSize;					
			SET_PHOTONS_DATA(hfcfg.Nphotons, hfcfg.mcx_pxSize, hfcfg.photonSize);

			/*Calculate memory settings*/
			memoryCalculator(&hfcfg, &nChunks, &photonsPerChunk, &remainingPhotons);
	
			/* Process photons struct in chunks if necessary*/
			photonOffset = 0;
			for (int chunk = 0; chunk < nChunks; chunk++) {
				photonOffset = photonsPerChunk * chunk * cfg->photonSize;
				if (chunk == nChunks - 1)
					load_photonStruct(&hfcfg, gpu, remainingPhotons, (photonsData + photonOffset), &photons);
				else
					load_photonStruct(&hfcfg, gpu, photonsPerChunk, (photonsData + photonOffset), &photons);
				printf("Running focusing kernel\n");
				for (int z = 0; z < cfg->nzPlanes; z++) {
					zFocus = ((cfg->zScan.y - cfg->zScan.x) / cfg->nzPlanes) * z + cfg->zScan.x;
					FOCUS_AT(zFocus / cfg->mcx_pxSize, z);
					//printf("Focus distance in grid units = %f\n", zFocus / hfcfg.mcx_pxSize);
					printf("\tFocusing at plane %d of %d. z= %f mm. \n", z + 1, cfg->nzPlanes, zFocus);
					focus4F << <gridSize, blockSize >> > (photons, gimageVolume);
					hipDeviceSynchronize();
				}

				CUDA_ASSERT(hipFree(photons));
			}
			free(photonsData);
			photonsData = NULL;
		}
		else
			printf("Error loading data\n");

	}
	printf("Focusing step finished\n");
	printf("Bringing images back to the host memory...\n");
	//build_photonStruct(cfg->Nphotons, photonsData, &photons, &photonsOrigin);	

	//printf("Running kernel for %d photons. %d blocks of %d threads per block\n", cfg->Nphotons, gridSize, blockSize);
	if (cfg->spimVol == 0) {
		volumeAdd(cfg, gpu, gimageVolume);		
		*imageVolume = (float*)calloc(cfg->sensorSize.x * cfg->sensorSize.y * 1, sizeof(float));
		CUDA_UNLOAD(imageVolume, gimageVolume, cfg->sensorSize.x * cfg->sensorSize.y * 1,float);
	}
	else {	
		*imageVolume = (float*)calloc(cfg->sensorSize.x * cfg->sensorSize.y * cfg->nzPlanes, sizeof(float));
		CUDA_UNLOAD(imageVolume, gimageVolume, cfg->sensorSize.x * cfg->sensorSize.y * cfg->nzPlanes,float);
	}
	
	printf("NA was %f\n", hfcfg.lens1R / (hfcfg.f1));

	//CUDA_ASSERT(hipFree(photons));
	return 0;
}

int focus_mcx(SPIMConfig* cfg, SPIMGPUInfo* gpu, photon* photons, float** imageVolume) {
    focusConfig hfcfg;
    //photon* photons;
    //photon0Pos* photonsOrigin;
    //float* photonsData = NULL;

    int nChunks;
    unsigned int photonsPerChunk, remainingPhotons;
    size_t cudaMemory;
    size_t cudaFreeMemory;
    float* gimageVolume;
    float zFocus;
    /* Prepare GPU settings*/
    int blockSize = 1024;				/**< The launch configurator returned block size*/
    int gridSize = (cfg->Nphotons) / blockSize + 1;	/**< The actual grid size needed, based on input size*/
    /* Prepare variables to import data files*/
    //int fidx = 0;
    char tmp[200];
    const char* dataFile;
    int Nphotons;
    unsigned int *gfocused;
    int focused;

    CUDA_ASSERT(hipMemGetInfo(&cudaFreeMemory, &cudaMemory));
    printf("Free memory space in GPU = %u KB = %.2f GB of %.2f GB\n", cudaFreeMemory, cudaFreeMemory / 1073741824.0f, cudaMemory / 1073741824.0f);

    /*Prepare camera buffer(instead of bulding a new sensor, we load buffer from host into GPU)*/
    CUDA_LOAD(&gimageVolume, *imageVolume, cfg->sensorSize.x * cfg->sensorSize.y * cfg->sensorSize.z,float);

   // build_cameraSensor(cfg, &gimageVolume);

    /*Generate kernel focusing settings*/
    createfocusConfig(cfg, gpu, &hfcfg);
    CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(fcfg), &hfcfg, sizeof(focusConfig)));

    /*Initialize photon stats*/
    CUDA_LOAD(&gfocused, cfg->focusedPhotons, cfg->sensorSize.z, unsigned int);

    //CUDA_ASSERT(hipMalloc((void**)& gfocused, sizeof(int)));
    //CUDA_ASSERT(hipMemset(gfocused, 0, sizeof(int)));
	  

    /*Update focusing kernel structure*/
    hfcfg.Nphotons = (unsigned long int)cfg->Nphotons;
    hfcfg.mcx_pxSize = cfg->mcx_pxSize;
    hfcfg.photonSize = cfg->photonSize;
    SET_PHOTONS_DATA(hfcfg.Nphotons, hfcfg.mcx_pxSize, hfcfg.photonSize);

    //print_photonData << < 1,1 >> > (photons, 30);
    prepare_MCXphotonData << <gridSize, blockSize >> > (photons);
    hipDeviceSynchronize();
    //print_photonData << < 1, 1 >> > (photons, 30);


    /*Run focusing kernel for each zPlane*/
    for (int z = 0; z < cfg->nzPlanes; z++) {
	zFocus = ((cfg->zScan.y - cfg->zScan.x) / cfg->nzPlanes) * z + cfg->zScan.x;
	FOCUS_AT(zFocus / cfg->mcx_pxSize, z * cfg->spimVol);/** SpimVol=1 for SPIM, 0 for OPT*/
	//printf("Focus distance in grid units = %f\n", zFocus / hfcfg.mcx_pxSize);
	printf("\tFocusing at plane %d of %d. z= %f mm.\n", z + 1, cfg->nzPlanes, zFocus);
	focus4F_auto << <gridSize, blockSize >> > (photons, gimageVolume,gfocused);
	CUDA_ERROR_CHECK(hipDeviceSynchronize());
	CUDA_UNLOAD(&(cfg->focusedPhotons), gfocused, cfg->sensorSize.z, unsigned int);
	//CUDA_ASSERT(hipMemcpy(&focused, gfocused, sizeof(int), hipMemcpyDeviceToHost));
	//printf("%e of %e photons used\n",(float)focused,(float)cfg->Nphotons);

    }
    
    printf("Focusing step finished\n");
    printf("Bringing images back to the host memory...\n");
    //build_photonStruct(cfg->Nphotons, photonsData, &photons, &photonsOrigin);	
    CUDA_UNLOAD(&(cfg->focusedPhotons),gfocused, cfg->sensorSize.z, unsigned int);

    //printf("Running kernel for %d photons. %d blocks of %d threads per block\n", cfg->Nphotons, gridSize, blockSize);
    CUDA_UNLOAD(imageVolume, gimageVolume, cfg->sensorSize.x * cfg->sensorSize.y * cfg->sensorSize.z,float);
    /*if (cfg->spimVol == 0) {
	//volumeAdd(cfg, gpu, gimageVolume);
	//*imageVolume = (float*)calloc(cfg->sensorSize.x * cfg->sensorSize.y * 1, sizeof(float));
	CUDA_UNLOAD(imageVolume, gimageVolume, cfg->sensorSize.x * cfg->sensorSize.y * 1);
    }
    else {
	//*imageVolume = (float*)calloc(cfg->sensorSize.x * cfg->sensorSize.y * cfg->nzPlanes, sizeof(float));
	CUDA_UNLOAD(imageVolume, gimageVolume, cfg->sensorSize.x * cfg->sensorSize.y * cfg->nzPlanes);
    }*/
    CUDA_ASSERT(hipFree(gfocused));
    CUDA_ASSERT(hipFree(gimageVolume));
    CUDA_ASSERT(hipFree(fcfg->illVolume));

    return 1;
}

/**
 * @brief Utility function to calculate the GPU stream processors (cores) per SM
 *
 * Obtain GPU core number per MP, this replaces
 * ConvertSMVer2Cores() in libcudautils to avoid
 * extra dependency.
 *
 * @param[in] v1: the major version of an NVIDIA GPU
 * @param[in] v2: the minor version of an NVIDIA GPU
 */
int corecount(int v1, int v2) {
	int v = v1 * 10 + v2;
	if (v < 20)      return 8;
	else if (v < 21) return 32;
	else if (v < 30) return 48;
	else if (v < 50) return 192;
	else if (v < 60) return 128;
	else          return 64;
}

/**
 * @brief Utility function to calculate the maximum blocks per SMX
 *
 * @param[in] v1: the major version of an NVIDIA GPU
 * @param[in] v2: the minor version of an NVIDIA GPU
 */
int smxblock(int v1, int v2) {
	int v = v1 * 10 + v2;
	if (v < 30)      return 8;
	else if (v < 50) return 16;
	else          return 32;
}

/**
 * @brief Utility function to query GPU info and set active GPU
 *
 * This function query and list all available GPUs on the system and print
 * their parameters. This is used when -L or -I is used.
 *
 * @param[in,out] cfg: the simulation configuration structure
 * @param[out] info: the GPU information structure
 */

int list_gpu(SPIMConfig*cfg, SPIMGPUInfo**info) {


	int dev;
	int deviceCount, activedev = 0;

	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		PRINT(stderr, "No CUDA-capable GPU device found\n");
		return 0;
	}
	*info = (SPIMGPUInfo*)calloc(deviceCount, sizeof(SPIMGPUInfo));
	if (cfg->gpuid && cfg->gpuid > deviceCount) {
		PRINT(stderr, "Specified GPU ID is out of range\n");
		return 0;
	}
	// scan from the first device
	for (dev = 0; dev < deviceCount; dev++) {
		hipDeviceProp_t dp;
		CUDA_ASSERT(hipGetDeviceProperties(&dp, dev));

		if (cfg->isgpuinfo == 3)
			activedev++;
		else if (cfg->deviceid[dev] == '1') {
			cfg->deviceid[dev] = '\0';
			cfg->deviceid[activedev] = dev + 1;
			activedev++;
		}
		strncpy((*info)[dev].name, dp.name, MAX_SESSION_LENGTH);
		(*info)[dev].id = dev + 1;
		(*info)[dev].devcount = deviceCount;
		(*info)[dev].major = dp.major;
		(*info)[dev].minor = dp.minor;
		(*info)[dev].globalmem = dp.totalGlobalMem;
		(*info)[dev].constmem = dp.totalConstMem;
		(*info)[dev].sharedmem = dp.sharedMemPerBlock;
		(*info)[dev].regcount = dp.regsPerBlock;
		(*info)[dev].clock = dp.clockRate;
		(*info)[dev].sm = dp.multiProcessorCount;
		(*info)[dev].core = dp.multiProcessorCount*corecount(dp.major, dp.minor);
		(*info)[dev].maxmpthread = dp.maxThreadsPerMultiProcessor;
		//(*info)[dev].maxgate = cfg->maxgate;
		(*info)[dev].autoblock = (*info)[dev].maxmpthread / smxblock(dp.major, dp.minor);
		(*info)[dev].autothread = (*info)[dev].autoblock * smxblock(dp.major, dp.minor) * (*info)[dev].sm;

		if (strncmp(dp.name, "Device Emulation", 16)) {
			if (cfg->isgpuinfo) {
				PRINT(stdout, "=============================   GPU Infomation  ================================\n");
				PRINT(stdout, "Device %d of %d:\t\t%s\n", (*info)[dev].id, (*info)[dev].devcount, (*info)[dev].name);
				PRINT(stdout, "Compute Capability:\t%u.%u\n", (*info)[dev].major, (*info)[dev].minor);
				PRINT(stdout, "Global Memory:\t\t%u B\nConstant Memory:\t%u B\n"
					"Shared Memory:\t\t%u B\nRegisters:\t\t%u\nClock Speed:\t\t%.2f GHz\n",
					(unsigned int)(*info)[dev].globalmem, (unsigned int)(*info)[dev].constmem,
					(unsigned int)(*info)[dev].sharedmem, (unsigned int)(*info)[dev].regcount, (*info)[dev].clock*1e-6f);
#if CUDART_VERSION >= 2000
				PRINT(stdout, "Number of MPs:\t\t%u\nNumber of Cores:\t%u\n",
					(*info)[dev].sm, (*info)[dev].core);
				PRINT(stdout, "Optimized blocks to launch:\t%u\nOptimized threads to launch:\t%u\n",
					(*info)[dev].autoblock, (*info)[dev].autothread);
#endif
				PRINT(stdout, "SMX count:\t\t%u\n", (*info)[dev].sm);
				PRINT(stdout, "================================================================================\n\n");
			}
		}
	}
	//if (cfg->isgpuinfo == 2 && cfg->parentid == mpStandalone) { //list GPU info only
	//	exit(0);
	//}
	if (activedev < MAX_DEVICE)
		cfg->deviceid[activedev] = '\0';
	return activedev;
}
