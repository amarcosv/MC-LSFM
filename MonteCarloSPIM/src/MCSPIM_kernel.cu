#include "hip/hip_runtime.h"
#include "MCSPIM_kernel.h"
#include "mcx_utils.h"
#include "mcx_core.h"
#include "focus_utils.h"
#include "focusKernel.h"
#include "string.h"
#include <time.h>
//--session line --root D:\Results\MonteCarlo_LSFM\Spiral\test_set\line -f D:\Results\MonteCarlo_LSFM\Spiral\test_set\line\vesselTest.json --outputformat mc2 --gpu 1 --photon 1470000000 --normalize 1 --save2pt 1 --reflect 0 --savedet 1 --unitinmm 0.10 --srcfrom0 1 --seed 1648335518 --saveseed 0 --specular 0 --array 0 --dumpmask 0 --repeat 1 -w XVWL --maxdetphoton 100000000 --bc aaaaaa
#define MU "\u03BC"

#define CUDA_ERROR_CHECK

#define CUDA_ASSERT( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
inline void __cudaSafeCall(hipError_t err, const char* file, const int line)
{
#ifdef CUDA_ERROR_CHECK
    if (hipSuccess != err)
    {
	fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
	    file, line, hipGetErrorString(err));
	exit(-1);
    }
#endif

    return;
}

/**
*@brief Error Handler for Kernel calls
*/
#define CUDA_ERROR_CHECK()    __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaCheckError(const char* file, const int line)
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
	fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
	    file, line, hipGetErrorString(err));
	exit(-1);
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if (hipSuccess != err)
    {
	fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
	    file, line, hipGetErrorString(err));
	exit(-1);
    }
#endif

    return;
}

/**
*@brief prints photon position information
*/
__device__ void print_photon_id(photon* photons,uint id) {
    printf("Printing photon #%d\n", id);
    printf("\t pos: [%f, %f, %f]\n", photons[id].pos.x, photons[id].pos.y, photons[id].pos.z);
    printf("\t dir: [%f, %f, %f]\n", photons[id].dir.x, photons[id].dir.y, photons[id].dir.z);
    printf("\t pos0: [%f, %f, %f]\n", photons[id].Opos.x, photons[id].Opos.y, photons[id].Opos.z);
    printf("\t w:	 %e\n", photons[id].w);

}

/**
*@brief prints photon position information
*/
__global__ void print_photon_id_cpu(photon* photons, uint id) {
    printf("Printing photon #%d\n", id);
    printf("\t pos: [%f, %f, %f]\n", photons[id].pos.x, photons[id].pos.y, photons[id].pos.z);
    printf("\t dir: [%f, %f, %f]\n", photons[id].dir.x, photons[id].dir.y, photons[id].dir.z);
    printf("\t pos0: [%f, %f, %f]\n", photons[id].Opos.x, photons[id].Opos.y, photons[id].Opos.z);
    printf("\t w:	 %e\n", photons[id].w);

}

/**
*@brief prints photon position information
*/
__device__ void print_photonPosition(float* photons, int index) {
    printf("Photon %d: pos[%f]\n", index, photons[index]);
    printf("Photon %d: pos[%f]\n", index+1, photons[index+1]);
    printf("Photon %d: pos[%f]\n", index+2, photons[index+2]);
    printf("Photon %d: pos[%f]\n", index+3, photons[index+3]);
    printf("Photon %d: pos[%f]\n", index+4, photons[index+4]);
    printf("Photon %d: pos[%f]\n", index+5, photons[index+5]);
    printf("Photon %d: pos[%f]\n", index+7, photons[index+7]);
    printf("Photon %d: pos[%f]\n", index+8, photons[index+8]);
    printf("Photon %d: pos[%f]\n", index+9, photons[index+9]);
    printf("Photon %d: pos[%f]\n", index + 6, photons[index + 6]);
}

/**
*@brief prints photon position information
*/
__global__ void print_photonPosition_cpu(float* photons, int index) {
    printf("Photon %d: pos[%f]\n", index, photons[index]);
    printf("Photon %d: pos[%f]\n", index + 1, photons[index + 1]);
    printf("Photon %d: pos[%f]\n", index + 2, photons[index + 2]);
    printf("Photon %d: pos[%f]\n", index + 3, photons[index + 3]);
    printf("Photon %d: pos[%f]\n", index + 4, photons[index + 4]);
    printf("Photon %d: pos[%f]\n", index + 5, photons[index + 5]);
    printf("Photon %d: pos[%f]\n", index + 7, photons[index + 7]);
    printf("Photon %d: pos[%f]\n", index + 8, photons[index + 8]);
    printf("Photon %d: pos[%f]\n", index + 9, photons[index + 9]);
    printf("Photon %d: pos[%f]\n", index + 6, photons[index + 6]);
}

/**
*@brief Initializes focus config structure from MCX parameters
*/
void merge_ConfigFiles(MCXConfig* mcxcfg, SPIMConfig* spimcfg) {
    spimcfg->workingDir = (char*)malloc(strlen(mcxcfg->rootpath));
    strcpy(spimcfg->workingDir, mcxcfg->rootpath);
    spimcfg->fileBaseName = (char*)malloc(strlen(mcxcfg->session));
    strcpy(spimcfg->fileBaseName, mcxcfg->session);
    spimcfg->mcx_pxSize = mcxcfg->unitinmm;
    spimcfg->mcx_simVolume = mcxcfg->dim;
    spimcfg->mua = mcxcfg->prop[1].mua; /** Comes in grid units*/
}

void initstats(MCSPIMStats* stats) {
    stats->simPhotons = 0;
    stats->detPhotons = 0;
    stats->mcxTime = 0;
    stats->focusTime = 0;
    stats->simTime = 0;
 }

void print_runningstats(MCSPIMStats* stats, SPIMConfig* spimconfig) {

    printf("**********************************************************************************\n\n");
    for (int i = 0; i < spimconfig->sensorSize.z; i++)
	printf("\t z Plane %d of %d. z= %f mm . Focused photons was %e\n", i + 1, spimconfig->sensorSize.z, ((spimconfig->zScan.y - spimconfig->zScan.x) / spimconfig->nzPlanes) * i + spimconfig->zScan.x, (float)stats->focusedPhotons[i]);
    printf("Total simulated photons was %e\n", stats->simPhotons);
    printf("Total detected photons was %e\n", stats->detPhotons);
    printf("Execution time: \n");
    printf("\tTotal: %f secs = %d hh %d mm %d ss\n", stats->simTime, (int)stats->simTime / 3600, ((int)stats->simTime % 3600) / 60, ((int)stats->simTime % 3600) % 60);
    printf("\tMCX:   %f secs \n", stats->mcxTime);
    printf("\tFocus: %f secs \n", stats->focusTime);
    printf("**********************************************************************************\n\n");
}

int save_runingLog(MCSPIMStats* stats, SPIMConfig* spimconfig, MCXConfig* mcxconfig) {
	FILE* fileID;
	size_t written;
	char idx[5];
	int fidx = 0;
	const char* dataFile;	
	char tmp[200];
	strcpy(tmp, spimconfig->workingDir);
	//strcat(tmp, "\\");
	strcat(tmp, spimconfig->outputFileName);

	strcat(tmp, ".txt");
	dataFile = &(*tmp);

	/** Get current time and date*/
	time_t t;   
	time(&t);

	printf("Writting results in file: %s\n", tmp);

	fileID = fopen(dataFile, "wb");
	if (NULL == fileID)
	    return -1; 
	fprintf(fileID,"***********************\tMC SPIM Simulation\t***********************\n");
	fprintf(fileID, "%s\n", ctime(&t));
	fprintf(fileID, "***********************\tData files and directories\t***********************\n");
	fprintf(fileID, "Output file\n");
	fprintf(fileID, "\tName = %s.mcspim\n", spimconfig->outputFileName + 1);
	fprintf(fileID, "\tDimensions [x y z] = [%u %u %u] px \n", spimconfig->sensorSize.x, spimconfig->sensorSize.x, spimconfig->sensorSize.z);
	if (spimconfig->spimVol == 1)
	    fprintf(fileID, "\tVoxel size = [%f %f %f] mm\n", (spimconfig->f2 / spimconfig->f1) * spimconfig->sensorpxSize, (spimconfig->f2 / spimconfig->f1) * spimconfig->sensorpxSize, (spimconfig->zScan.y - spimconfig->zScan.x) / (float)spimconfig->nzPlanes);
	else
	    fprintf(fileID, "\Pixel size = %f mm\n", (spimconfig->f2 / spimconfig->f1)*spimconfig->sensorpxSize); 	
	fprintf(fileID, "Data directories\n");
	fprintf(fileID, "\tBasename = %s\n", spimconfig->fileBaseName);
	fprintf(fileID, "\tWorking directory = %s\n", spimconfig->workingDir);
	fprintf(fileID, "\tConfig file = %s\n", spimconfig->configFileName+1);
	fprintf(fileID, "\tIllumination file name = %s\n", spimconfig->illFileName+1);

	fprintf(fileID, "\n***********************\tSimulation parameters\t***********************\n");
	fprintf(fileID, "Number of simulations = %u\n", spimconfig->nFiles);
	fprintf(fileID, "Optical properties:\n");
	fprintf(fileID, "\t%sa = %f 1/cm\n", MU, 10.f * mcxconfig->prop[1].mua / mcxconfig->unitinmm);
	fprintf(fileID, "\t%ss = %f 1/cm\n", MU, 10.f * mcxconfig->prop[1].mus / mcxconfig->unitinmm);
	fprintf(fileID, "\tg = %f\n", mcxconfig->prop[1].g);
	fprintf(fileID, "\tn = %f\n", mcxconfig->prop[1].n);
	fprintf(fileID, "Fluorescence:\n");
	fprintf(fileID, "\tSource type = %d\n", mcxconfig->srctype);
	if(mcxconfig->srctype==16)
	fprintf(fileID, "\tSource file = %s\n", mcxconfig->fluoname);
	fprintf(fileID, "\tNumber of fluorophores = %u\n",(unsigned int) mcxconfig->srcparam1.x);

	fprintf(fileID, "MCX configuration:\n");
	fprintf(fileID, "\tVolume size = [%u %u %u] px\n", spimconfig->mcx_simVolume.x, spimconfig->mcx_simVolume.y, spimconfig->mcx_simVolume.z);
	fprintf(fileID, "\tPixel size = %f\n", spimconfig->mcx_pxSize);
	fprintf(fileID, "Illumination configuration:\n");
	fprintf(fileID, "\tVolume size = [%u %u %u] px\n", spimconfig->ill_simVolume.x, spimconfig->ill_simVolume.y, spimconfig->ill_simVolume.z);
	fprintf(fileID, "\tPixel size = %f\n", spimconfig->ill_pxSize);
	fprintf(fileID, "\tDetailed illumination = %s\n", (spimconfig->ill_simVolume.y == 1) ? "Yes" : "No");
		
	fprintf(fileID, "Camera configuration:\n");
	fprintf(fileID, "\tSensor size  = [%u %u] px = [%f %f] mm\n", spimconfig->sensorSize.x, spimconfig->sensorSize.y, ((float)spimconfig->sensorSize.x) * spimconfig->sensorpxSize, ((float)spimconfig->sensorSize.y) * spimconfig->sensorpxSize);
	fprintf(fileID, "\tSensor pixel size = %f mm\n", spimconfig->sensorpxSize);
	fprintf(fileID, "\tFocal length [f1 f2] = [%.2f %.2f] mm \n", spimconfig->f1, spimconfig->f2);
	fprintf(fileID, "\tNumerical aperture [NA1 NA2] = [%.2f %.2f] \n", spimconfig->NA1, spimconfig->NA2);	
	fprintf(fileID, "SPIM Acquisition:\n");
	fprintf(fileID, "\tZ scanning range [init end zstep] = [%.2f %.2f %.4f] mm\n", spimconfig->zScan.x, spimconfig->zScan.y, (spimconfig->zScan.y- spimconfig->zScan.x)/(float)spimconfig->nzPlanes);
	fprintf(fileID, "\tZ planes  = %u\n", spimconfig->nzPlanes);
	fprintf(fileID, "\tImage mode  = %s\n", ((spimconfig->spimVol==1) ? "SPIM" : "OPT"));

	fprintf(fileID, "\n***********************\tRunning stats\t***********************\n");
	fprintf(fileID, "Simulated photons: \n");
	fprintf(fileID, "\tTotal = %e\n", stats->simPhotons);
	fprintf(fileID, "\tDetected = %e\n", stats->detPhotons);
	fprintf(fileID, "\tDetection rate = %.2f%%\n", stats->detPhotons/stats->simPhotons*100);
	fprintf(fileID, "Execution time: \n");
	fprintf(fileID, "\tTotal: %f secs = %d hours %d mins %d secs\n", stats->simTime, (int)stats->simTime / 3600, ((int)stats->simTime % 3600) / 60, ((int)stats->simTime % 3600) % 60);
	fprintf(fileID, "\tMCX:   %f secs \n", stats->mcxTime);
	fprintf(fileID, "\tFocus: %f secs \n", stats->focusTime);
	fprintf(fileID, "Focused photons per plane : \n");
	for (int i = 0; i < spimconfig->sensorSize.z; i++)
	    fprintf(fileID, "\t z Plane %d of %d. z= %f mm . Focused photons was %e\n", i + 1, spimconfig->sensorSize.z, ((spimconfig->zScan.y - spimconfig->zScan.x) / spimconfig->nzPlanes) * i + spimconfig->zScan.x, (float)stats->focusedPhotons[i]);

	fclose(fileID);
	printf("File written succesfully!\n");
	return 0;

}

/**
*@brief Launches mcx simulation and MCfocusing code
*/
int simulationLauncher(int argc, char* argv[]) {

    /*! structure to store all simulation parameters
    */
    SPIMConfig spimconfig;	    /**< spimconfig: structure to store all focusing algorithm parameters */
    MCXConfig  mcxconfig;           /**< mcxconfig: structure to store all simulation parameters */
    GPUInfo* gpuinfo = NULL;        /**< gpuinfo: structure to store GPU information */
    SPIMGPUInfo* spimgpuinfo = NULL;    /**< Structure for GPU configuration */
    MCSPIMStats stats;		    /**< stats: structure with all execution stats*/

    float* outputVolume;	    /**< outputVolume: buffer to store output volume */
    float* imageVolume;		    /**< imageVolume: buffer to store spim volume */
    float* illVolume = NULL;	    /**< illVolume: buffer to store illuminationVolume */
    unsigned int activedev = 0;     /**< activedev: count of total active GPUs to be used */
    
    size_t cudaMemory;			/**<  Cuda total device memory*/
    size_t cudaFreeMemory;		/**<  Cuda device free memory*/
    size_t imageVolumeSize;		/**<  Output volume size*/
    size_t illVolumeSize;		/**<  Illumination volume size*/
    size_t cudaFreememory_ph;	/**<  Cuda device free memory in photons*/
  
    size_t Nphotons;			/**<  Photons to run at each simulation*/
    float* gPdet;			/**< GPU pointer to MCX detected photons array*/
    photon* detphotons;			/**< GPU pointer to casted detected photons */

    /** Variables for running statistics*/
    float simPhotons = 0;		/**< Total simulated photons*/
    float detPhotons = 0;		/**< Total detected photons*/   
    float mcxTime = 0;			/**< Total MCX simulation time*/   
    float focusTime = 0;		/**< Total focusing time*/  
    float simTime = 0;			/**< Total simulation time*/
    clock_t begin, end;
    clock_t simStart, simEnd;

    simStart = clock();

    /**
       To start an MCX simulation, we first create a simulation configuration and
       set all elements to its default settings.
     */
    mcx_initcfg(&mcxconfig);
    initcfg(&spimconfig);
    initstats(&stats);

    /**
       Then, we parse the full command line parameters and set user specified settings
     */
    mcx_parsecmd(argc, argv, &mcxconfig);
    merge_ConfigFiles(&mcxconfig, &spimconfig);
    readConfigFile(&spimconfig);

    /** The next step, we identify gpu number and query all GPU info */
    if (!(activedev = mcx_list_gpu(&mcxconfig, &gpuinfo))) {
	mcx_error(-1, "No GPU device found\n", __FILE__, __LINE__);
    }
    if (!(list_gpu(&spimconfig, &spimgpuinfo))) {
	printf("no active GPU device found\n");
    }
    /**
	Specify here compulsary settings for a MCX dedicated MCSPIM simulation
    */
    Nphotons = mcxconfig.nphoton;
    mcxconfig.savedetflag = 180;
    //mcxconfig.nphoton= 1000;
    /**
	Prepare output data here
    */
    outputVolume = (float *)calloc(spimconfig.sensorSize.x * spimconfig.sensorSize.y * spimconfig.nzPlanes,sizeof(float));
    spimconfig.focusedPhotons = (unsigned int*)calloc(spimconfig.sensorSize.z, sizeof(unsigned int));
    stats.focusedPhotons = spimconfig.focusedPhotons;

    /**Perform memory precalculations to ensure enough memory  */
    CUDA_ASSERT(hipMemGetInfo(&cudaFreeMemory, &cudaMemory));
    imageVolumeSize = spimconfig.sensorSize.x * spimconfig.sensorSize.y * spimconfig.sensorSize.z * sizeof(float);
    illVolumeSize = spimconfig.ill_simVolume.x * spimconfig.ill_simVolume.y * spimconfig.ill_simVolume.z*sizeof(float);
    cudaFreememory_ph = ((cudaFreeMemory - (imageVolumeSize + illVolumeSize)) / sizeof(photon)) * 0.97;

    imageVolume=(float*)calloc(spimconfig.sensorSize.x * spimconfig.sensorSize.y * spimconfig.sensorSize.z, sizeof(float));
    /**
       This line runs the main MCX simulation for each GPU inside each thread
     */
  
    
    printf("Loading illumination file in memory.......\n");

    //	printf("Error loading data\n");
    int result;
    result = importIlluminationData(&spimconfig, &illVolume);
    if (result == 0)
	printf("Illumination was loaded succesfully!!\n");
    else {
	printf("ERROR: Illumination file couldn't be loaded.\n");
	printf("Exiting the program....\n");
	return 1;
    }

    for (int i = 0; i < spimconfig.nFiles; i++) {
	printf("Running simulation %d of %d...\n", i + 1,spimconfig.nFiles);
	CUDA_ASSERT(hipMemGetInfo(&cudaFreeMemory, &cudaMemory));
	printf("Initially free memory space in GPU = %u KB = %.2f GB of %.2f GB\n", cudaFreeMemory, cudaFreeMemory / 1073741824.0f, cudaMemory / 1073741824.0f);
	mcxconfig.nphoton= Nphotons ;
	mcxconfig.maxdetphoton = (unsigned int)cudaFreememory_ph;

	begin = clock();
	mcx_run_simulation(&mcxconfig, gpuinfo, &gPdet);
	end = clock();

	if (mcxconfig.detectedcount > mcxconfig.maxdetphoton)
	    spimconfig.Nphotons = mcxconfig.maxdetphoton;
	else
	    spimconfig.Nphotons = mcxconfig.detectedcount;
	
	stats.detPhotons = stats.detPhotons + (float)spimconfig.Nphotons;
	stats.simPhotons = stats.simPhotons + (float)mcxconfig.nphoton;
	stats.mcxTime = stats.mcxTime + (float)(end - begin) / CLOCKS_PER_SEC;

	detphotons = (photon*)gPdet;
	printf("MCX Simulation ended after running %e photons\n", (float)mcxconfig.nphoton);
	printf("\Detected %e photons of a maximum of %e\n", (float)spimconfig.Nphotons, (float)mcxconfig.maxdetphoton);
	printf("\nUsing %f %% of photon memory\n", ((float)spimconfig.Nphotons / (float)mcxconfig.maxdetphoton) * 100.0f);

	/*Prepare camera buffer*/
        //build_cameraSensor(cfg, &gimageVolume);

	begin = clock();
	focus_mcx(&spimconfig, spimgpuinfo, detphotons, &imageVolume);
	end = clock();

	stats.focusTime= stats.focusTime + (float)(end - begin) / CLOCKS_PER_SEC;

	detphotons = NULL;
	CUDA_ASSERT(hipFree(gPdet));

	//update_outputVolume(&spimconfig, spimgpuinfo, &outputVolume, imageVolume); We reuse our buffer all the time

	exportVolume(imageVolume, &spimconfig);
	if (result == 0)
	    printf("Ouput was exported succesfully!!\n");
	else
	    printf("Error writting data file\n");
	/**
	   Once simulation is complete, we clean up the allocated memory in config and gpuinfo, and exit
	 */
	mcxconfig.seed = mcxconfig.seed + 1;
	CUDA_ASSERT(hipDeviceReset());

    }
    simEnd = clock();
    stats.simTime = (float)(simEnd - simStart) / CLOCKS_PER_SEC;

    save_runingLog(&stats, &spimconfig, &mcxconfig);
    print_runningstats(&stats, &spimconfig);



    mcx_cleargpuinfo(&gpuinfo);
    mcx_clearcfg(&mcxconfig);
    return 0;


}